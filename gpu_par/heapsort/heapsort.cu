#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "../common/cuPrintf.cu"

#define BLOCKSIZE 1023 //Size of blocks at the bottom heap
#define METASIZE 511   //Size of metaheap
#define METACACHE 4    //Size of metacache
#define METADEPTH 9    //Max depth of metaheap (ceil of log of metaSize)
#define OUTSIZE 512    //Size of output shared memory
#define BLOCKDEPTH 10  //Max Depth of bottom heap, and ceil of log of blocksize
#define MINWARPS 1     //Minimum warp count to run code.
#define INVALID -1     //special value signifying invalid Buffer/heap entry.

typedef struct metaEntry {
    float value;
    short key;
} metaEntry_t;


//Tells us our current progress on building a given block.
typedef struct blockInfo{
    int bufSize;    //How many popped elements are buffered right now?
    int writeLoc;   //Index into blockwrite array for next write
    short remaining;  //How many elements are left to pop?
    short size;       //Total number of elements in the block.
    short index;      //Which block are we?
    short heapified;  //Only a bool is needed, but short maintains alignment.
} blockInfo_t;


//Forward declarations
__global__ void GPUHeapSort(float *d_list, float *midList, float *sortedList,
                            blockInfo_t *blockInfo,
                            int numBlocks, int len, int topHeapSize,
                            int botHeapSize, 
                            int warpSize, int metaDepth);
__device__ void bottomLevel(float *d_list, int len); //NYI
__device__ void topLevel(float *d_list, int len); //NYI
__device__ void heapify(__volatile__ float *in_list, int len);
__device__ void pipelinedPop(__volatile__ float *heap, float *out_list, 
                             int d, int popCount);
__device__ void loadBlock(float *g_block, float *s_block,
                          blockInfo_t *g_info, blockInfo_t *s_info);
__device__ void writeBlock(float *g_block, float *s_block,
                           int writeLen,
                           blockInfo_t *g_info, blockInfo_t *s_info);
__device__ void printBlock(float *s_block, int blockLen);
__device__ void initBlocks(blockInfo_t *blockInfo, int numBlocks, int len);
__device__ void initMetaHeap(metaEntry *heap, float BUF[METASIZE][METACACHE]);
__host__ int heapSort(float *h_list, 
                      int len, int threadsPerBlock,
                      int blocks, hipDeviceProp_t devProp);
__host__ int floorlog2(int x);

//Ceiling of log2 of x.  Could be made faster, but effect would be negligible.
int ceilLog2(int x){
    if (x < 1){
        return -1;
    }
    x--;
    int output = 0;
    while (x > 0) {
        x >>= 1;
        output++;
    }
    return output;
}

/* Heapsort definition.  Takes a pointer to a list of floats.
 * the length of the list, the number of threads per block, and 
 * the number of blocks on which to execute.  
 * Puts the list into sorted order in-place.*/
int heapSort(float *h_list, int len, int threadsPerBlock, int blocks,
              hipDeviceProp_t devProp) {

    float *d_list, *midList, *sortedList; //various lists that will live on GPU
    blockInfo_t *blockInfo;
    int logLen; //log of length of list
    int metaDepth; //layers of metaheaps
    int topHeapSize; //Size of the top heap
    int logBotHeapSize; //log_2 of max size of the bottom heaps 
    int logMidHeapSize; //log_2 of max size of intermediate heaps
    int numBlocks; //Number of bottom heaps.  Poor choice of name =p.
    int temp;

    //Trivial list?  Just return.
    if (len < 2){
        return 0;
    }

    //Ensure that we have a valid number of threads per block.
    if (threadsPerBlock == 0){
        threadsPerBlock = devProp.maxThreadsPerBlock;
    }
    //We require a minimum of 2 warps per block to run our code
    else if (threadsPerBlock < 2*devProp.warpSize){
        printf("At least 2 warps are required to run heapsort.  ");
        printf("Increasing thread count to 64.\n");
        threadsPerBlock = MINWARPS*devProp.warpSize;
    }
    if (threadsPerBlock > devProp.maxThreadsPerBlock) {
        printf("Device cannot handle %d threads per block.  Max is %d\n",
               threadsPerBlock, devProp.maxThreadsPerBlock);
        return -1;
    }
    //We require a minimum of 2 blocks to run our code.
    if (blocks < 2){
        printf("At least 2 blocks are required to run heapsort.\n");
        return -1;
    }
     
    //Calculate size of heaps.  BotHeapSize is 1/8 shared mem size.
    //logBotHeapSize = ceilLog2(devProp.sharedMemPerBlock>>3);
    logBotHeapSize = BLOCKDEPTH;
    logMidHeapSize = logBotHeapSize - 2;

    printf("logBotHeap: %d, logMidHeap: %d\n", logBotHeapSize, logMidHeapSize);

    //Calculate metaDepth and topHeapSize.
    metaDepth = 0; //Will increment this if necessary.
    logLen = ceilLog2(len);
    temp = logBotHeapSize; //temp is a counter tracking total subheap depth.
    
    //Do we only need one heap?
    if (temp >= logLen){
        topHeapSize = len;
    }
    //Otherwise, how many metaheaps do we need?
    else {
        while (temp < logLen){
            metaDepth++;
            temp += logMidHeapSize;
        }
        topHeapSize = len>>temp;
    }

    printf("metaDepth is %d\n", metaDepth);
    printf("topHeapSize is %d\n", topHeapSize); 

    if (metaDepth > blocks){
        printf("Must have at least metaDepth blocks available.");
        printf("metaDepth is %d, but only %d blocks were given.\n", 
               metaDepth, blocks);
        return -1;
    }

    if (metaDepth > 2){
        printf("Current implementation only supports metaDepth of 2.  ");
        printf("Given metadepth was %d.  In practice, ", metaDepth); 
        printf("this means that list lengths cannot equal or exceed 2^20.");
    }


    if ( (hipMalloc((void **) &d_list, len*sizeof(float))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return -1;
    }

    if ( (hipMalloc((void **) &midList, len*sizeof(float))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return -1;
    }

    if ( (hipMalloc((void **) &sortedList, len*sizeof(float))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return -1;
    }

    hipMemcpy(d_list, h_list, len*sizeof(float), hipMemcpyHostToDevice);
    
    if ( (hipMalloc((void **) &blockInfo, len*sizeof(blockInfo_t))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return -1;
    }
    
    numBlocks = ceil((float)len/BLOCKSIZE); //number of bottom heaps
    printf("numHBlocks: %d\n", numBlocks);

    printf("Attempting to call GPUHeapSort\n\n");
    /*
    GPUHeapSort<<<blocks, threadsPerBlock,
        ( 1<<(logBotHeapSize + 3) + 1<<(logBotHeapSize+2) ) >>>
        (d_list, blockInfo, len, topHeapSize, 1<<logBotHeapSize, 
         devProp.warpSize, metaDepth);
    */
    GPUHeapSort<<<blocks, threadsPerBlock>>>
        (d_list, midList, sortedList, blockInfo, numBlocks, 
         len, 0, BLOCKSIZE, devProp.warpSize, metaDepth);

    hipDeviceSynchronize();
    hipMemcpy(h_list, d_list, len*sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}

/* GPUHeapSort definition.  Takes a pointer to a list of floats, the length
 * of the list, and the number of list elements given to each thread.
 * Puts the list into sorted order in-place.*/
__global__ void GPUHeapSort(float *d_list, float *midList, float *sortedList,
                            blockInfo_t *blockInfo,
                            int numBlocks, int len, int topHeapSize,
                            int botHeapSize,
                            int warpSize, int metaDepth){
    
    
    if (blockIdx.x == 0) { //NYI

        /*
        __shared__ metaEntry heap[METASIZE];
        __shared__ float buffer[METASIZE][METACACHE];
        __shared__ float output[OUTSIZE];
        __shared__ blockInfo_t curBlockInfo;
        */
        //Initialize datastructures
        //initMetaHeap(heap, buffer);

    }
    else {

        __shared__ float heap[BLOCKSIZE];
        __shared__ float output[OUTSIZE];
        __shared__ blockInfo_t curBlockInfo;
        __shared__ int curIdx;
        __shared__ int popCount; //How many heap elements are we popping?

        cuPrintf("About to call init\n");
        
        //Initialize datastructures
        initBlocks(blockInfo, numBlocks, len);
        cuPrintf("Init finished.\n");
        __syncthreads();
        
        curIdx = blockIdx.x-1;
        
        while (curIdx < numBlocks){
            //Load memory
            loadBlock(&d_list[curIdx*BLOCKSIZE], (float *)heap,
                      &blockInfo[curIdx], &curBlockInfo);
            
            cuPrintf("curBlockInfo:  (bufsize: %d, writeloc: %d, heapified: %d\
 remaining: %d, size: %d\n",
                     curBlockInfo.bufSize, curBlockInfo.writeLoc,
                     curBlockInfo.heapified, curBlockInfo.remaining,
                     curBlockInfo.size);
            
            if (curBlockInfo.heapified == 0){
                //First warp heapifies
                if (threadIdx.x < 8){
                    heapify(heap, curBlockInfo.size);
                }
                curBlockInfo.heapified = 1;
                __syncthreads();
            }
            cuPrintf("Entering while Loop\n");
            while (curBlockInfo.remaining > 0){
                //First warp pops
                
                cuPrintf("curBlockInfo:  (bufsize: %d, writeloc: %d, heapified: %d\
 remaining: %d, size: %d\n",
                         curBlockInfo.bufSize, curBlockInfo.writeLoc,
                         curBlockInfo.heapified, curBlockInfo.remaining,
                         curBlockInfo.size);
                
                
                if (threadIdx.x == 0){
                    popCount = curBlockInfo.remaining;
                    if (popCount > OUTSIZE) {
                        popCount = OUTSIZE;
                    }
                    curBlockInfo.remaining -= popCount;
                }
                if (threadIdx.x < 8){
                    pipelinedPop(heap, (float *)output, BLOCKDEPTH, popCount);
                }
                
                __syncthreads();
                cuPrintf("Calling writeBlock with popcount %d\n", popCount);
                
                writeBlock(d_list, output, popCount,
                           &blockInfo[curBlockInfo.index], &curBlockInfo);
                __syncthreads();
                cuPrintf("At end, remaining:  %d\n", curBlockInfo.remaining);
            }
            cuPrintf("After the while loop...\n");
        
        curIdx += (gridDim.x - 1);
        }
    }
    return;

}

/* Loads a block of data from global memory into shared memory.  Must be
 * called by all threads of a thread block to ensure proper operation.
 * g_info:  A pointer to the specific (global) blockinfo to be read. 
 */
__device__ void loadBlock(float *g_block, float *s_block,
                          blockInfo_t *g_info, blockInfo_t *s_info){
    
    if (threadIdx.x == 0){
        *s_info = *g_info;
        __threadfence_block();
    }

    cuPrintf("Entering loadBlock\n");
    for(int i = threadIdx.x; i < BLOCKSIZE; i += gridDim.x){
        if(i < s_info->size){
            s_block[i] = g_block[i];
        } 
        else {
            s_block[i] = 0;
        }
    }
    __syncthreads();
    return;
}

/* Writes a block of data from shared memory into global memory.  Must be
 * called by all threads of a thread block to ensure proper operation. 
 * g_info:  A pointer to the specific (global) g_info to be written.
 */
__device__ void writeBlock(float *g_block, float *s_block, int writeLen,
                           blockInfo_t *g_info, blockInfo_t *s_info){
    cuPrintf("beginning writeBlock\n");
    for(int i = threadIdx.x; i < writeLen; i += blockDim.x){
        g_block[s_info->writeLoc+i] = s_block[i];
        cuPrintf("writing block %d to %d with value %f\n",
                 i, s_info->writeLoc + i, g_block[s_info->writeLoc + i]);
    }
    __syncthreads();
    //Update the blockInfo struct in global memory
    if (threadIdx.x == 0){
        //s_info->writeLoc += writeLen;
        //*g_info = *s_info;
        cuPrintf("setting writeLoc to %d\n", s_info->writeLoc + writeLen);
        s_info->writeLoc += writeLen;
        g_info->writeLoc = s_info->writeLoc;
        g_info->remaining = s_info->remaining;
        atomicAdd(&g_info->bufSize, writeLen);
    }
    __syncthreads();
    return;
}

/* Prints a block of data in shared memory */
__device__ void printBlock(float *s_block, int blockLen){
    for (int i = threadIdx.x; i < blockLen; i += blockDim.x){
        cuPrintf("s_block[%d] = %f\n", i, s_block[i]);
    }
}

/* Initializes data structures for heapsort.  Must be run by all threads
 * of all blocks.
 * blockInfo:  A pointer to the entire array of blockInfos.
 */
__device__ void initBlocks(blockInfo_t *blockInfo, int numBlocks, int len){

    if ((threadIdx.x == 0) && (blockIdx.x != 0) ){

        cuPrintf("attempting to init\n");
        //Initialize blockinfo structs.  Initialization is done by the blocks
        //that own each blockinfo struct.
        blockInfo_t BI;
        BI.bufSize = 0;
        BI.heapified = 0;
        BI.remaining = BLOCKSIZE;
        BI.size = BLOCKSIZE;
        for (int idx = (blockIdx.x-1); idx < numBlocks; idx += (gridDim.x-1)){
            BI.writeLoc = idx*BLOCKSIZE;
            BI.index = idx;
            cuPrintf("writeloc is %d\n", BI.writeLoc);
            //Did we overrun our bounds when setting size?
            if ((idx+1)*BLOCKSIZE > len){
                BI.size = len - idx*BLOCKSIZE;
                BI.remaining = BI.size;
            }
            blockInfo[idx] = BI;
        }
    }
    __syncthreads();
}

__device__ void initMetaHeap(metaEntry *heap, float buf[METASIZE][METACACHE]){

    for (int i = threadIdx.x; i < METASIZE; i += blockDim.x){
        heap[i].value = INVALID;
        heap[i].key = i;
        for (int j = 0; j < METACACHE; j++){
            buf[i][j] = INVALID;
            cuPrintf("Invalidating buf[%d][%d]\n", i, j); 
        }
    }

    return;
}

/* Heapifies a list using a single warp.  Must be run on the bottom warp of a
 * thread.  If this function is not executed by all of threads 0-7, the GPU
 * will stall.
 */
__device__ void heapify(__volatile__ float *inList, int len){
    
    int focusIdx = 0; //Index of element currently being heapified
    float focus=0, parent=0; //current element being heapified and its parent
    __volatile__ __shared__ int temp;
    /*int localTemp=0; Temp doesn't need to be re-read _every_ time.
                    * Temp will be used to track the next element to percolate.
                    */

    if (threadIdx.x == 0){
        temp = 0; //Index of next element to heapify
    }

    //localTemp = 0;
    
    //We maintain the invariant that no two threads are processing on
    //adjacent layers of the heap in order to avoid memory conflicts and
    //race conditions.
    while (temp < len){
        if (threadIdx.x == (temp & 7)){
            focusIdx = temp;
            focus = inList[focusIdx];
            temp = temp + 1;
            //cuPrintf("Focusing on element %d with value %f\n",
            //         focusIdx, focus);
        }
        
        //Unrolled loop once to avoid race conditions and get a small speed
        //boost over using a for loop on 2 iterations.
        if (focusIdx != 0){
            parent = inList[(focusIdx-1)>>1];
            //Swap focus and parent if focus is bigger than parent
            if (focus > parent){
                //cuPrintf("Focus %f > parent %f\n", focus, parent); 
                inList[focusIdx] = parent;
                inList[(focusIdx-1)>>1] = focus;
                focusIdx = (focusIdx - 1)>>1;
            }
            else {
                //cuPrintf("Parent %f > focus %f\n", parent, focus);
                focusIdx = 0;
            }
        }
        if (focusIdx != 0){
            parent = inList[(focusIdx-1)>>1];
            //Swap focus and parent if focus is bigger than parent
            if (focus > parent){
                //cuPrintf("Focus %f > parent %f\n", focus, parent); 
                inList[focusIdx] = parent;
                inList[(focusIdx-1)>>1] = focus;
                focusIdx = (focusIdx-1)>>1;
            }
            else {
                //cuPrintf("Parent %f > focus %f\n", parent, focus);
                focusIdx = 0; 
            }
       }
        //localTemp = *temp;
    }
    
    //Empty the pipeline before returning
    while (focusIdx !=0){
        parent = inList[(focusIdx-1)>>1];
        //Swap focus and parent if focus is bigger than parent
        if (focus > parent){
            cuPrintf("Focus %f > parent %f\n", focus, parent); 
            inList[focusIdx] = parent;
            inList[(focusIdx-1)>>1] = focus;
            focusIdx = (focusIdx-1)>>1;
        }
        else {
            //cuPrintf("Parent %f > focus %f\n", parent, focus);
            focusIdx = 0; 
        }
    }
    
    return;
}

/* Pops a heap using a single warp.  Must be run on the bottom warp of a
 * thread.  If this function is not executed by all of threads 0-7, the GPU
 * will stall.
 * heap: a pointer to a heap structure w/ space for a complete heap of depth d 
 * d:  The depth of the heap 
 * count: The number of elements to pop
 */
__device__ void pipelinedPop(__volatile__ float *heap, float *out_list, 
                             int d, int popCount){
    
    int focusIdx = 0; //Index of element currently percolating down
    int maxChildIdx=0; //Index of largest child of element percolating down
    int curDepth=d+1; //Depth of element currently percolating down
    __volatile__ __shared__ int temp;
    /*int localTemp=0; Temp doesn't need to be re-read _every_ time.
                    * Temp will be used to track the next element to percolate.
                    */

    if (threadIdx.x == 0){
        temp = 0; //We have thus far popped 0 elements
    }

    //localTemp = 0;
    
    //We maintain the invariant that no two threads are processing on
    //adjacent layers of the heap in order to avoid memory conflicts and
    //race conditions.
    while (temp < popCount){
        if (threadIdx.x == (temp & 7)){
            focusIdx = 0;
            curDepth = 0;
            out_list[temp] = heap[0];
            temp = temp + 1;
            //cuPrintf("temp is: %d\n", *temp);
            //cuPrintf("top of heap is: %f\n", heap[0]);
        }
        
        //Unrolled loop once to avoid race conditions and get a small speed
        //boost over using a for loop on 2 iterations.
        if (curDepth < d-1){
            maxChildIdx = 2*focusIdx+1;
            //cuPrintf("Children are %f, %f\n", heap[2*focusIdx+2], 
            //         heap[maxChildIdx]); 
            //cuPrintf("Depth is %d, Focusing on element %d\n", curDepth,
            //         focusIdx);
            if (heap[2*focusIdx+2] > heap[maxChildIdx]){
                maxChildIdx = 2*focusIdx+2;
            }
            heap[focusIdx] = heap[maxChildIdx];
            focusIdx = maxChildIdx;
            curDepth++;
        }

        if (curDepth < d-1){
            maxChildIdx = 2*focusIdx+1;
            //cuPrintf("Depth is %d, Focusing on element %d\n", curDepth,
            //         focusIdx);
            if (heap[2*focusIdx+2] > heap[maxChildIdx]){
                maxChildIdx = 2*focusIdx+2;
            }
            heap[focusIdx] = heap[maxChildIdx];
            focusIdx = maxChildIdx;
            curDepth++;
        }

        if (curDepth == d-1){
            //cuPrintf("curDepth is %d\n", curDepth);
            //cuPrintf("focusIdx is %d\n", focusIdx);
            //cuPrintf("Depth is %d (max).  Focusing on element %d\n", curDepth,
            //focusIdx);
            heap[focusIdx] = 0;
            curDepth++;
            //continue;
        }
    }
    
    //empty the pipeline before returning
    
    while (curDepth < d-1){
        //cuPrintf("Emptying Pipeline.  Focusing on element %d\n", focusIdx); 
        maxChildIdx = 2*focusIdx+1;
        if (heap[2*focusIdx+2] > heap[maxChildIdx]){
            maxChildIdx = 2*focusIdx+2;
        }
        heap[focusIdx] = heap[maxChildIdx];
        focusIdx = maxChildIdx;
        curDepth++;
    }
    if (curDepth == d-1){
        //cuPrintf("curDepth is %d\n", curDepth);
        //cuPrintf("focusIdx is %d\n", focusIdx);
        //cuPrintf("Depth is %d (max).  Focusing on element %d\n", curDepth,
        //focusIdx);
        heap[focusIdx] = 0;
        curDepth++;
        //continue;
    }

    return;
}

void usage(){
    printf("Usage: in_list [thread_count] [kernel_count]\n"); 
}

int main(int argc, char *argv[]){
    
    int len;
    float *h_list;

    cudaPrintfInit();

    if ((argc > 4) || argc < 2) {
        printf("Invalid argument count.  %s accepts 1-4 arguments, %d given\n",
               argv[0], argc);
        usage();
        return -1;
    }
    
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    int thread_count = 64;
    //int block_count = devProp.maxGridSize[0];
    int block_count = 2;

    if (argc > 2){
        thread_count = atoi(argv[2]);
    }
    if (argc > 3){
        block_count = atoi(argv[3]);
    }

    FILE *fin = fopen(argv[1], "r");
    
    if (fin == NULL){
        printf("Could not open file: %s", argv[1]);
        return -2;
    }

    fscanf(fin, "%d", &len);

    h_list = (float *)malloc(len*sizeof(float));
    if (h_list == NULL){
        printf("Insufficient host memory to allocate at %d", __LINE__);
        return -3;
    }

    for (int i = 0; i < len; i++){
        if (EOF == fscanf(fin, "%f ", &h_list[i])){
            break;
        }
    }

    /*
    printf("\nInitial list is:\n");
    for (int i = 0; i < len; i++){
        printf("%f\n", h_list[i]);
    }
    */

    //MergeSort(h_list, len, devProp.maxThreadsDim[0], devProp.maxGridSize[0]);
    //MergeSort(h_list, len, devProp.maxThreadsDim[0], 1);
    heapSort(h_list, len, thread_count, block_count, devProp);

    hipDeviceSynchronize();
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    
    printf("\nFinal list is:\n");
    for (int i = 0; i < len; i++){
        printf("%f\n", h_list[i]);
    }
    

    return 0;
}
