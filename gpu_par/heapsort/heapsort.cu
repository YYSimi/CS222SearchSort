#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "../common/cuPrintf.cu"

#define BLOCKSIZE 1023  //Size of blocks at the bottom heap
#define OUTSIZE 512 //Size of output shared memory
#define BLOCKDEPTH 10 //Max Depth of bottom heap, and ceil of log of blocksize

//Tells us our current progress on building a given block.
typedef struct blockInfo{
    short bufSize;    //How many popped elements are buffered right now?
    short writeLoc;   //Index into blockwrite array for next write
    short remaining;  //How many elements are left to pop?
    short size;       //Total number of elements in the block.
    short heapified;  //Only a bool is needed, but short maintains alignment.
} blockInfo_t;


//Forward declarations
__global__ void GPUHeapSort(float *d_list, float *midList, float *sortedList,
                            blockInfo_t *blockInfo,
                            int numBlocks, int len, int topHeapSize,
                            int botHeapSize, 
                            int warpSize, int metaDepth);
__device__ void bottomLevel(float *d_list, int len); //NYI
__device__ void topLevel(float *d_list, int len); //NYI
__device__ void heapify(__volatile__ float *in_list, int len);
__device__ void pipelinedPop(__volatile__ float *heap, float *out_list, 
                             int d, int popCount);
__device__ void loadBlock(float *g_block, float *s_block, int readLen,
                          blockInfo_t *g_info, blockInfo_t *s_info);
__device__ void writeBlock(float *g_block, float *s_block,
                           int writeLen,
                           blockInfo_t *g_info, blockInfo_t *s_info);
__device__ void printBlock(float *s_block, int blockLen);
__device__ void init(blockInfo_t *blockInfo, int numBlocks, int len);
__host__ int heapSort(float *h_list, 
                      int len, int threadsPerBlock,
                      int blocks, hipDeviceProp_t devProp);
__host__ int floorlog2(int x);

//Ceiling of log2 of x.  Could be made faster, but effect would be negligible.
int ceilLog2(int x){
    if (x < 1){
        return -1;
    }
    x--;
    int output = 0;
    while (x > 0) {
        x >>= 1;
        output++;
    }
    return output;
}

/* Heapsort definition.  Takes a pointer to a list of floats.
 * the length of the list, the number of threads per block, and 
 * the number of blocks on which to execute.  
 * Puts the list into sorted order in-place.*/
int heapSort(float *h_list, int len, int threadsPerBlock, int blocks,
              hipDeviceProp_t devProp) {

    float *d_list, *midList, *sortedList; //various lists that will live on GPU
    blockInfo_t *blockInfo;
    int logLen; //log of length of list
    int metaDepth; //layers of metaheaps
    int topHeapSize; //Size of the top heap
    int logBotHeapSize; //log_2 of max size of the bottom heaps 
    int logMidHeapSize; //log_2 of max size of intermediate heaps
    int numBlocks; //Number of bottom heaps.  Poor choice of name =p.
    int temp;

    //Trivial list?  Just return.
    if (len < 2){
        return 0;
    }

    //Ensure that we have a valid number of threads per block.
    if (threadsPerBlock == 0){
        threadsPerBlock = devProp.maxThreadsPerBlock;
    }
    //We require a minimum of 2 warps per block to run our code
    else if (threadsPerBlock < 2*devProp.warpSize){
        printf("At least 2 warps are required to run heapsort.  ");
        printf("Increasing thread count to 64.\n");
        threadsPerBlock = 64;
    }
    if (threadsPerBlock > devProp.maxThreadsPerBlock) {
        printf("Device cannot handle %d threads per block.  Max is %d\n",
               threadsPerBlock, devProp.maxThreadsPerBlock);
        return -1;
    }
    //We require a minimum of 2 blocks to run our code.
    if (blocks < 2){
        printf("At least 2 blocks are required to run heapsort.\n");
        return -1;
    }
     
    //Calculate size of heaps.  BotHeapSize is 1/8 shared mem size.
    //logBotHeapSize = ceilLog2(devProp.sharedMemPerBlock>>3);
    logBotHeapSize = BLOCKDEPTH;
    logMidHeapSize = logBotHeapSize - 2;

    printf("logBotHeap: %d, logMidHeap: %d\n", logBotHeapSize, logMidHeapSize);

    //Calculate metaDepth and topHeapSize.
    metaDepth = 0; //Will increment this if necessary.
    logLen = ceilLog2(len);
    temp = logBotHeapSize; //temp is a counter tracking total subheap depth.
    
    //Do we only need one heap?
    if (temp >= logLen){
        topHeapSize = len;
    }
    //Otherwise, how many metaheaps do we need?
    else {
        while (temp < logLen){
            metaDepth++;
            temp += logMidHeapSize;
        }
        topHeapSize = len>>temp;
    }

    printf("metaDepth is %d\n", metaDepth);
    printf("topHeapSize is %d\n", topHeapSize); 

    if (metaDepth > blocks){
        printf("Must have at least metaDepth blocks available.");
        printf("metaDepth is %d, but only %d blocks were given.\n", 
               metaDepth, blocks);
        return -1;
    }

    if (metaDepth > 2){
        printf("Current implementation only supports metaDepth of 2.  ");
        printf("Given metadepth was %d.  In practice, ", metaDepth); 
        printf("this means that list lengths cannot equal or exceed 2^20.");
    }


    if ( (hipMalloc((void **) &d_list, len*sizeof(float))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return -1;
    }

    if ( (hipMalloc((void **) &midList, len*sizeof(float))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return -1;
    }

    if ( (hipMalloc((void **) &sortedList, len*sizeof(float))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return -1;
    }

    hipMemcpy(d_list, h_list, len*sizeof(float), hipMemcpyHostToDevice);
    
    if ( (hipMalloc((void **) &blockInfo, len*sizeof(blockInfo_t))) == 
         hipErrorOutOfMemory) {
        printf("Error:  Insufficient device memory at line %d\n", __LINE__);
        return -1;
    }
    
    numBlocks = ceil((float)len/BLOCKSIZE); //number of bottom heaps
    printf("numHBlocks: %d\n", numBlocks);

    printf("Attempting to call GPUHeapSort\n\n");
    /*
    GPUHeapSort<<<blocks, threadsPerBlock,
        ( 1<<(logBotHeapSize + 3) + 1<<(logBotHeapSize+2) ) >>>
        (d_list, blockInfo, len, topHeapSize, 1<<logBotHeapSize, 
         devProp.warpSize, metaDepth);
    */
    GPUHeapSort<<<blocks, threadsPerBlock>>>
        (d_list, midList, sortedList, blockInfo, numBlocks, 
         len, 0, BLOCKSIZE, devProp.warpSize, metaDepth);

    hipDeviceSynchronize();
    hipMemcpy(h_list, d_list, len*sizeof(float), hipMemcpyDeviceToHost);

    return 0;
}

/* GPUHeapSort definition.  Takes a pointer to a list of floats, the length
 * of the list, and the number of list elements given to each thread.
 * Puts the list into sorted order in-place.*/
__global__ void GPUHeapSort(float *d_list, float *midList, float *sortedList,
                            blockInfo_t *blockInfo,
                            int numBlocks, int len, int topHeapSize,
                            int botHeapSize,
                            int warpSize, int metaDepth){
    
    __shared__ float heap[BLOCKSIZE];
    __shared__ float output[OUTSIZE];
    __shared__ blockInfo_t curBlockInfo;
    __shared__ int g_start, g_end;
    __shared__ int blockLen;
    __shared__ int popCount; //How many heap elements are we popping?

    cuPrintf("My ID is %d\n", blockIdx.x);
    if (blockIdx.x == 0) { //NYI
        //cuPrintf("Block 0 reporting in\n");
    }
    else {
        cuPrintf("Uh?");
    
        //Initialize datastructures
        init(blockInfo, numBlocks, len);
        __syncthreads();

        g_start = (blockIdx.x-1)*botHeapSize;
        g_end = (blockIdx.x)*botHeapSize;    
        
        if (g_end > len){
            g_end = len;
        }
        
        blockLen = g_end-g_start;
        
        //Load memory
        loadBlock(&d_list[g_start], (float *)heap, blockLen,
                  &blockInfo[blockIdx.x-1], &curBlockInfo);
        
        __syncthreads();
        
        cuPrintf("curBlockInfo:  (bufsize: %d, writeloc: %d, heapified: %d \
remaining: %d, size: %d\n",
                 curBlockInfo.bufSize, curBlockInfo.writeLoc,
                 curBlockInfo.heapified, curBlockInfo.remaining,
                 curBlockInfo.size);

        if (curBlockInfo.heapified == 0){
            //First warp heapifies
            if (threadIdx.x < 8){
                heapify(heap, blockLen);
            }
            curBlockInfo.heapified = 1;
            __syncthreads();
        }
        cuPrintf("Entering while Loop\n");
        while (curBlockInfo.remaining > 0){
            //First warp pops
            if (threadIdx.x == 0){
                popCount = curBlockInfo.remaining;
                if (popCount > OUTSIZE) {
                    popCount = OUTSIZE;
                }
                curBlockInfo.remaining -= popCount;
            }
            if (threadIdx.x < 8){
                pipelinedPop(heap, (float *)output, BLOCKDEPTH, popCount);
            }
            
            //writeBlock
            
            __syncthreads();
            cuPrintf("Calling writeBlock with popcount %d\n", popCount);
            writeBlock(&d_list[g_start], output, popCount,
                       &blockInfo[blockIdx.x-1], &curBlockInfo);
            __syncthreads();
        }
    }
    return;

}

/* Loads a block of data from global memory into shared memory.  Must be
 * called by all threads of a thread block to ensure proper operation. 
 */
__device__ void loadBlock(float *g_block, float *s_block, int readLen,
                          blockInfo_t *g_info, blockInfo_t *s_info){
    
    for(int i = threadIdx.x; i < readLen; i += blockDim.x){
        s_block[i] = g_block[i]; 
    }
    if (threadIdx.x == 0){
        *s_info = g_info[blockIdx.x - 1];
    }
    return;
}

/* Writes a block of data from shared memory into global memory.  Must be
 * called by all threads of a thread block to ensure proper operation. 
 */
__device__ void writeBlock(float *g_block, float *s_block, int writeLen,
                           blockInfo_t *g_info, blockInfo_t *s_info){
    cuPrintf("beginning writeBlock\n");
    for(int i = threadIdx.x; i < writeLen; i += blockDim.x){
        g_block[s_info->writeLoc+i] = s_block[i];
        cuPrintf("writing block %d to %d with value %f\n",
                 s_info->writeLoc + i, i, s_block[i]);
    }
    __syncthreads();
    //Update the blockInfo struct in global memory
    if (threadIdx.x == 0){
        s_info->writeLoc += writeLen;
        *g_info = *s_info;
    }

    return;
}

/* Prints a block of data in shared memory */
__device__ void printBlock(float *s_block, int blockLen){
    for (int i = threadIdx.x; i < blockLen; i += blockDim.x){
        cuPrintf("s_block[%d] = %f\n", i, s_block[i]);
    }
}

/* Initializes data structures for heapsort.  Must be run by all threads
 * of all blocks. */
__device__ void init(blockInfo_t *blockInfo, int numBlocks, int len){
    cuPrintf("wtf mate?\n");
    if ((threadIdx.x == 0) && (blockIdx.x != 0) ){

        cuPrintf("attempting to init\n");
        //Initialize blockinfo structs.  Initialization is done by the blocks
        //that own each blockinfo struct.
        blockInfo_t BI;
        BI.bufSize = 0;
        BI.heapified = 0;
        BI.remaining = BLOCKSIZE;
        BI.size = BLOCKSIZE;
        for (int idx = (blockIdx.x-1); idx < numBlocks; idx += (blockDim.x-1)){
            BI.writeLoc = idx*BLOCKSIZE;
            cuPrintf("writeloc is %d\n", BI.writeLoc);
            //Did we overrun our bounds when setting size?
            if ((idx+1)*BLOCKSIZE > len){
                BI.size = len - idx*BLOCKSIZE;
                BI.remaining = BI.size;
            }
            blockInfo[idx] = BI;
            cuPrintf("BI stored size: %d\n", blockInfo[idx].size); 
        }
    }
    __syncthreads();
}

/* Heapifies a list using a single warp.  Must be run on the bottom warp of a
 * thread.  If this function is not executed by all of threads 0-7, the GPU
 * will stall.
 */
__device__ void heapify(__volatile__ float *inList, int len){
    
    int focusIdx = 0; //Index of element currently being heapified
    float focus=0, parent=0; //current element being heapified and its parent
    __volatile__ __shared__ int temp;
    /*int localTemp=0; Temp doesn't need to be re-read _every_ time.
                    * Temp will be used to track the next element to percolate.
                    */

    if (threadIdx.x == 0){
        temp = 0; //Index of next element to heapify
    }

    //localTemp = 0;
    
    //We maintain the invariant that no two threads are processing on
    //adjacent layers of the heap in order to avoid memory conflicts and
    //race conditions.
    while (temp < len){
        if (threadIdx.x == (temp & 7)){
            focusIdx = temp;
            focus = inList[focusIdx];
            temp = temp + 1;
            //cuPrintf("Focusing on element %d with value %f\n",
            //         focusIdx, focus);
        }
        
        //Unrolled loop once to avoid race conditions and get a small speed
        //boost over using a for loop on 2 iterations.
        if (focusIdx != 0){
            parent = inList[(focusIdx-1)>>1];
            //Swap focus and parent if focus is bigger than parent
            if (focus > parent){
                //cuPrintf("Focus %f > parent %f\n", focus, parent); 
                inList[focusIdx] = parent;
                inList[(focusIdx-1)>>1] = focus;
                focusIdx = (focusIdx - 1)>>1;
            }
            else {
                //cuPrintf("Parent %f > focus %f\n", parent, focus);
                focusIdx = 0;
            }
        }
        if (focusIdx != 0){
            parent = inList[(focusIdx-1)>>1];
            //Swap focus and parent if focus is bigger than parent
            if (focus > parent){
                //cuPrintf("Focus %f > parent %f\n", focus, parent); 
                inList[focusIdx] = parent;
                inList[(focusIdx-1)>>1] = focus;
                focusIdx = (focusIdx-1)>>1;
            }
            else {
                //cuPrintf("Parent %f > focus %f\n", parent, focus);
                focusIdx = 0; 
            }
       }
        //localTemp = *temp;
    }
    return;
}

/* Pops a heap using a single warp.  Must be run on the bottom warp of a
 * thread.  If this function is not executed by all of threads 0-7, the GPU
 * will stall.
 * heap: a pointer to a heap structure w/ space for a complete heap of depth d 
 * d:  The depth of the heap 
 * count: The number of elements to pop
 */
__device__ void pipelinedPop(__volatile__ float *heap, float *out_list, 
                             int d, int popCount){
    
    int focusIdx = 0; //Index of element currently percolating down
    int maxChildIdx=0; //Index of largest child of element percolating down
    int curDepth=d+1; //Depth of element currently percolating down
    __volatile__ __shared__ int temp;
    /*int localTemp=0; Temp doesn't need to be re-read _every_ time.
                    * Temp will be used to track the next element to percolate.
                    */

    if (threadIdx.x == 0){
        temp = 0; //We have thus far popped 0 elements
    }

    //localTemp = 0;
    
    //We maintain the invariant that no two threads are processing on
    //adjacent layers of the heap in order to avoid memory conflicts and
    //race conditions.
    while (temp < popCount){
        if (threadIdx.x == (temp & 7)){
            focusIdx = 0;
            curDepth = 0;
            out_list[temp] = heap[0];
            temp = temp + 1;
            //cuPrintf("temp is: %d\n", *temp);
            //cuPrintf("top of heap is: %f\n", heap[0]);
        }
        
        //Unrolled loop once to avoid race conditions and get a small speed
        //boost over using a for loop on 2 iterations.
        if (curDepth < d-1){
            maxChildIdx = 2*focusIdx+1;
            //cuPrintf("Children are %f, %f\n", heap[2*focusIdx+2], 
            //         heap[maxChildIdx]); 
            //cuPrintf("Depth is %d, Focusing on element %d\n", curDepth,
            //         focusIdx);
            if (heap[2*focusIdx+2] > heap[maxChildIdx]){
                maxChildIdx = 2*focusIdx+2;
            }
            heap[focusIdx] = heap[maxChildIdx];
            focusIdx = maxChildIdx;
            curDepth++;
        }

        if (curDepth < d-1){
            maxChildIdx = 2*focusIdx+1;
            //cuPrintf("Depth is %d, Focusing on element %d\n", curDepth,
            //         focusIdx);
            if (heap[2*focusIdx+2] > heap[maxChildIdx]){
                maxChildIdx = 2*focusIdx+2;
            }
            heap[focusIdx] = heap[maxChildIdx];
            focusIdx = maxChildIdx;
            curDepth++;
        }

        if (curDepth == d-1){
            //cuPrintf("curDepth is %d\n", curDepth);
            //cuPrintf("focusIdx is %d\n", focusIdx);
            //cuPrintf("Depth is %d (max).  Focusing on element %d\n", curDepth,
            //focusIdx);
            heap[focusIdx] = 0;
            curDepth++;
            //continue;
        }
    }
    
    //empty the pipeline before returning
    
    while (curDepth < d-1){
        //cuPrintf("Emptying Pipeline.  Focusing on element %d\n", focusIdx); 
        maxChildIdx = 2*focusIdx+1;
        if (heap[2*focusIdx+2] > heap[maxChildIdx]){
            maxChildIdx = 2*focusIdx+2;
        }
        heap[focusIdx] = heap[maxChildIdx];
        focusIdx = maxChildIdx;
        curDepth++;
    }
    

    return;
}

void usage(){
    printf("Usage: in_list [thread_count] [kernel_count]\n"); 
}

int main(int argc, char *argv[]){
    
    int len;
    float *h_list;

    cudaPrintfInit();

    if ((argc > 4) || argc < 2) {
        printf("Invalid argument count.  %s accepts 1-4 arguments, %d given\n",
               argv[0], argc);
        usage();
        return -1;
    }
    
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    int thread_count = 64;
    //int block_count = devProp.maxGridSize[0];
    int block_count = 2;

    if (argc > 2){
        thread_count = atoi(argv[2]);
    }
    if (argc > 3){
        block_count = atoi(argv[3]);
    }

    FILE *fin = fopen(argv[1], "r");
    
    if (fin == NULL){
        printf("Could not open file: %s", argv[1]);
        return -2;
    }

    fscanf(fin, "%d", &len);

    h_list = (float *)malloc(len*sizeof(float));
    if (h_list == NULL){
        printf("Insufficient host memory to allocate at %d", __LINE__);
        return -3;
    }

    for (int i = 0; i < len; i++){
        if (EOF == fscanf(fin, "%f ", &h_list[i])){
            break;
        }
    }

    /*
    printf("\nInitial list is:\n");
    for (int i = 0; i < len; i++){
        printf("%f\n", h_list[i]);
    }
    */

    //MergeSort(h_list, len, devProp.maxThreadsDim[0], devProp.maxGridSize[0]);
    //MergeSort(h_list, len, devProp.maxThreadsDim[0], 1);
    heapSort(h_list, len, thread_count, block_count, devProp);

    hipDeviceSynchronize();
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    
    printf("\nFinal list is:\n");
    for (int i = 0; i < len; i++){
        printf("%f\n", h_list[i]);
    }
    

    return 0;
}
